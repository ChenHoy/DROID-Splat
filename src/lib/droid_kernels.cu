#include "hip/hip_runtime.h"
// Copyright (c) 2022 Copyright holder of the paper "DroidCalib: deep geometry-aware camera 
// self-calibration from video" submitted to "IEEE/CVF Conference on Computer Vision and 
// Pattern Recognition 2023" for review.
// All rights reserved.
//
// This source code is derived from DROID-SLAM (https://github.com/princeton-vl/DROID-SLAM)
// Copyright (c) 2021, Princeton Vision & Learning Lab, licensed under the BSD 3-Clause License,
// cf. 3rd-party-licenses.txt file in the root directory of this source tree.


#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
#include <math.h>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>

// #include "utils.cuh"

#include <Eigen/Sparse>
#include <Eigen/SparseCore>
#include <Eigen/SparseCholesky>

typedef Eigen::SparseMatrix<double> SpMat;
typedef Eigen::Triplet<double> T;
typedef std::vector<std::vector<long>> graph_t;
typedef std::vector<torch::Tensor> tensor_list_t;



#define MIN_DEPTH 0.25

#define THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + THREADS - 1) / THREADS)


#define GPU_1D_KERNEL_LOOP(k, n) \
  for (size_t k = threadIdx.x; k<n; k += blockDim.x)


__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

__device__ void blockReduce(volatile float *sdata) {
  unsigned int tid = threadIdx.x;
  __syncthreads();

  // if (threadIdx.x < 256) {sdata[tid] += sdata[tid + 256]; } __syncthreads();
  if (threadIdx.x < 128) {sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (threadIdx.x <  64) {sdata[tid] += sdata[tid +  64]; } __syncthreads();

  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();
}


__device__ void
actSO3(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const float *t, const float *q, const float *X, float *Y) {
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
adjSE3(const float *t, const float *q, const float *X, float *Y) {
  float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3(qinv, &X[0], &Y[0]);
  actSO3(qinv, &X[3], &Y[3]);

  float u[3], v[3];
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  actSO3(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

__device__ void 
relSE3(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

  actSO3(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

__device__ void
rotmat(const float *qij, float R[][3]){

  // storage of quaternion as x, y, z, w
  float s = 1.0; // unit quaternions

  R[0][0] = 1 - 2 * s * (qij[1]*qij[1] + qij[2]*qij[2]);
  R[0][1] = 2 * s * (qij[0]*qij[1] - qij[2]*qij[3]);
  R[0][2] = 2 * s * (qij[0]*qij[2] + qij[1]*qij[3]);

  R[1][0] = 2 * s * (qij[0]*qij[1] + qij[2]*qij[3]);
  R[1][1] = 1 - 2 * s * (qij[0]*qij[0] + qij[2]*qij[2]);
  R[1][2] = 2 * s * (qij[1]*qij[2] - qij[0]*qij[3]);

  R[2][0] = 2 * s * (qij[0]*qij[2] - qij[1]*qij[3]);
  R[2][1] = 2 * s * (qij[1]*qij[2] + qij[0]*qij[3]);
  R[2][2] = 1 - 2 * s * (qij[0]*qij[0] + qij[1]*qij[1]);

}

  
__device__ void
expSO3(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3(const float *xi, float* t, float* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  float tau[3] = {xi[0], xi[1], xi[2]};
  float phi[3] = {xi[3], xi[4], xi[5]};

  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    float a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    float b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}



__device__ void
proj_transform_mei(const float* uvi, const float* qij, const float* tij, 
        const float fx, const float fy, const float cx, const float cy, 
        const float xi, const float disp, float* uvj) {

  const float xs = (uvi[0] - cx) / fx;
  const float ys = (uvi[1] - cy) / fy;
  const float beta = (xi + sqrt( 1 + ( 1 - xi * xi) * (xs * xs  + ys * ys)) ) / (1 + xs * xs  + ys * ys);

  float Xi[4];
  float Xj[4];

  Xi[0] = beta * xs / (beta - xi);
  Xi[1] = beta * ys / (beta - xi);
  Xi[2] = 1.0;
  Xi[3] = disp;

  actSE3(tij, qij, Xi, Xj);

  const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
  const float z = (Xj[2] < MIN_DEPTH) ? 0.0 : Xj[2];
  const float r = sqrt(Xj[0] * Xj[0] + Xj[1] * Xj[1] + z * z);
  const float mei_fact = ((z + xi * r) < MIN_DEPTH) ? 0.0 : 1.0 / (z + xi * r);

  uvj[0] = fx * Xj[0] * mei_fact + cx ;
  uvj[1] = fy * Xj[1] * mei_fact + cy ;
}




__global__ void projective_transform_kernel(
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> Hs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> vs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eii,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eij,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Cii,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> bz,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Calib,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> CalibPose,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> CalibDepth,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> qs,
    const bool opt_intr,
    const int n_intr,
    const int model_id  // 0: pinhole, 1:mei
    )
{
  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  int ix = static_cast<int>(ii[block_id]);
  int jx = static_cast<int>(jj[block_id]);

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;
  __shared__ float xi;


  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4]; 
  __shared__ float R[3][3]; 

  if (thread_id == 0) {
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];

    if (model_id > 0){
      xi = intrinsics[4];
    }

  }

  __syncthreads();

  // stereo frames
  if (ix == jx) {
    if (thread_id == 0) {
      tij[0] =  -0.1;
      tij[1] =     0;
      tij[2] =     0;
      qij[0] =     0;
      qij[1] =     0;
      qij[2] =     0;
      qij[3] =     1;
    }
  }

  else {

    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();

    if (thread_id == 0) {
      relSE3(ti, qi, tj, qj, tij, qij);
      rotmat(qij, R);
    }

  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  // jacobians
  float Jx[12];
  float Jz;
  float Jc[5]; //n_intr=4 or 5; using 5 to have sufficient space

  float* Ji = &Jx[0];
  float* Jj = &Jx[6];

  // pose hessian
  float hij[12*(12+1)/2];

  // calib hessian
  float cij[5*(5+1)/2]; 
  
  // calib-pose-blcok 
  float cpij[2][5][6]; 
  float q[5]; 

  float vi[6], vj[6];

  // initialize everything with zero
  int l;
  for (l=0; l<12*(12+1)/2; l++) {
    hij[l] = 0;
  }

  for (l=0; l<2; l++) {
    for (int n=0; n<n_intr; n++) {
      for (int m=0; m<6; m++) {
        cpij[l][n][m] = 0;
      }
    }  
  }

  for (l=0; l<n_intr*(n_intr+1)/2; l++) {
    cij[l] = 0;
  }

  for (int n=0; n<6; n++) {
    vi[n] = 0;
    vj[n] = 0;
  }
  
  for (int n=0; n<n_intr; n++) {
    q[n] = 0;
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {

    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);

    float uvi[2];
    float uvj[2];
    uvi[0] = u;
    uvi[1] = v;

    float ru;
    float rv;

    if (model_id == 0){
      Xi[0] = (u - cx) / fx;
      Xi[1] = (v - cy) / fy;
      Xi[2] = 1;
      Xi[3] = disps[ix][i][j];
    }
    else {
      const float xs = (u - cx) / fx;
      const float ys = (v - cy) / fy;
      const float beta = (xi + sqrt( 1 + ( 1 - xi * xi) * (xs * xs  + ys * ys)) ) / (1 + xs * xs  + ys * ys);

      Xi[0] = beta * xs / (beta - xi);
      Xi[1] = beta * ys / (beta - xi);
      Xi[2] = 1;
      Xi[3] = disps[ix][i][j];
    }
    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    const float x = Xj[0];
    const float y = Xj[1];
    const float h = Xj[3];

    const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
    const float d2 = d * d;
    const float z = (Xj[2] < MIN_DEPTH) ? 0.0 : Xj[2];
    const float r = sqrt(x * x + y * y + z * z);
    

    float wu = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][0][i][j];
    float wv = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][1][i][j];

    if (model_id == 0){
      ru = target[block_id][0][i][j] - (fx * d * x + cx);
      rv = target[block_id][1][i][j] - (fy * d * y + cy);
    }
    else {
      const float mei_fact = ((z + xi * r) < MIN_DEPTH) ? 0.0 : 1.0 / (z + xi * r);
      ru = target[block_id][0][i][j] - (fx * x * mei_fact + cx);
      rv = target[block_id][1][i][j] - (fy * y * mei_fact + cy);
    }

    // x - coordinate
    if (model_id == 0){
      Jj[0] = fx * (h*d);
      Jj[1] = fx * 0;
      Jj[2] = fx * (-x*h*d2);
      Jj[3] = fx * (-x*y*d2);
      Jj[4] = fx * (1 + x*x*d2);
      Jj[5] = fx * (-y*d);
      Jz = fx * (tij[0] * d - tij[2] * (x * d2));
    }
    else {
      const float mei_fact = ((z + xi * r) < MIN_DEPTH) ? 0.0 : 1.0 / (z + xi * r);
      const float mei_fact2 = mei_fact * mei_fact;

      Jj[0] = h * (fx * mei_fact - fx * xi * x * x / r * mei_fact2);
      Jj[1] = -h * (fx * xi * x * y / r * mei_fact2);
      Jj[2] = -h * (fx * x * ((xi*z) / r + 1.0) * mei_fact2);

      Jj[3] = z * (fx * xi * x * y / r * mei_fact2) - y * (fx * x * ((xi*z) / r + 1.0) * mei_fact2);
      Jj[4] = z * (fx * mei_fact - fx * xi * x * x / r * mei_fact2) + x * (fx * x * ((xi*z) / r + 1.0) * mei_fact2);
      Jj[5] = -y * (fx * mei_fact - fx * xi * x * x / r * mei_fact2) - x * (fx * xi * x * y / r * mei_fact2);

      Jz = tij[0] * (fx * mei_fact - fx * xi * x * x / r * mei_fact2) - tij[1] * (fx * xi * x * y / r * mei_fact2) - tij[2] * (fx * x * ((xi*z) / r + 1.0) * mei_fact2) ;
    }

    Cii[block_id][k] = wu * Jz * Jz;
    bz[block_id][k] = wu * ru * Jz;

    if (ix == jx) wu = 0;


    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1;

    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += wu * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      vi[n] += wu * ru * Ji[n];
      vj[n] += wu * ru * Jj[n];

      Eii[block_id][n][k] = wu * Jz * Ji[n];
      Eij[block_id][n][k] = wu * Jz * Jj[n];
    }

    if (opt_intr){

      // calib jacobian pinhole
      if (model_id == 0){
        Jc[0] = -R[0][0] * (u-cx) * (1.0/fx) * d + R[2][0] * (u-cx) * (1.0/fx) * x * d2 + x * d;
        Jc[1] = -R[0][1] * (v-cy) * (1.0/fy) * (1.0/fy) * fx * d + R[2][1] * (v-cy) * (1.0/fy) * (1.0/fy) * fx * x * d2;
        Jc[2] = -R[0][0] * d + x * d2 * R[2][0] + 1.0;
        Jc[3] = -R[0][1] * fx * (1.0/fy) * d + fx * x * d2 * (1.0/fy) * R[2][1];
      }
      else{ 
        // calib jacobian mei model: numerical approximation because analytical formulas are pages long
        const float mu = 0.0001;
        const float disp = disps[ix][i][j];
        float u_low;
        float u_up;

        proj_transform_mei(uvi, qij, tij, fx-mu, fy, cx,  cy,  xi, disp, uvj);
        u_low = uvj[0];
        proj_transform_mei(uvi, qij, tij, fx+mu, fy, cx,  cy,  xi, disp, uvj);
        u_up = uvj[0];
        Jc[0] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy-mu, cx,  cy,  xi, disp, uvj);
        u_low = uvj[0];
        proj_transform_mei(uvi, qij, tij, fx, fy+mu, cx,  cy,  xi, disp, uvj);
        u_up = uvj[0];
        Jc[1] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy, cx-mu,  cy,  xi, disp, uvj);
        u_low = uvj[0];
        proj_transform_mei(uvi, qij, tij, fx, fy, cx+mu,  cy,  xi, disp, uvj);
        u_up = uvj[0];
        Jc[2] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy-mu,  xi, disp, uvj);
        u_low = uvj[0];
        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy+mu,  xi, disp, uvj);
        u_up = uvj[0];
        Jc[3] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy,  xi-0.01*mu, disp, uvj);
        u_low = uvj[0];
        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy,  xi+0.01*mu, disp, uvj);
        u_up = uvj[0];
        Jc[4] = (u_up - u_low) / (2 * 0.01 * mu);
      }
      
      // calib block
      l=0;
      for (int n=0; n<n_intr; n++) {
        for (int m=0; m<=n; m++) {
          cij[l] += wu * Jc[n] * Jc[m];
          l++;
        }
      }

      // calib-pose block
      for (int n=0; n<n_intr; n++) {
        for (int m=0; m<6; m++) {
          cpij[0][n][m] += wu * Jc[n] * Ji[m];
          cpij[1][n][m] += wu * Jc[n] * Jj[m];
        }
      }


      for (int n=0; n<n_intr; n++) {
        q[n] += wu * ru * Jc[n];
        CalibDepth[block_id][k][n] = wu * Jz * Jc[n];
      }

    }

    // y - coordinate
    if (model_id == 0){
      Jj[0] = fy * 0;
      Jj[1] = fy * (h*d);
      Jj[2] = fy * (-y*h*d2);
      Jj[3] = fy * (-1 - y*y*d2);
      Jj[4] = fy * (x*y*d2);
      Jj[5] = fy * (x*d);
      Jz = fy * (tij[1] * d - tij[2] * (y * d2));
    }
    else{
      const float mei_fact = ((z + xi * r) < MIN_DEPTH) ? 0.0 : 1.0 / (z + xi * r);
      const float mei_fact2 = mei_fact * mei_fact;
      Jj[0] =  - h * fy * xi * x * y / r * mei_fact2;
      Jj[1] =  h * ( fy * mei_fact - fy * xi * y * y / r * mei_fact2);
      Jj[2] =  -h * (fy * y * (xi * (z) / r + 1.0) * mei_fact2);
      Jj[3] =  -z * (fy * mei_fact - fy * xi * y * y / r * mei_fact2) - y * (fy * y * (xi * (z) / r + 1.0) * mei_fact2);
      Jj[4] =  -z * (fy * xi * x * y / r * mei_fact2) + x * (fy * y * (xi * (z) / r + 1.0) * mei_fact2);
      Jj[5] =  y * (fy * xi * x * y / r * mei_fact2) + x * ( fy * mei_fact - fy * xi * y * y / r * mei_fact2);
      Jz =  -tij[0] * (fy * xi * x * y / r * mei_fact2) + tij[1] * ( fy * mei_fact - fy * xi * y * y / r * mei_fact2) - tij[2] * (fy * y * (xi * (z) / r + 1.0) *mei_fact2);
    }

    Cii[block_id][k] += wv * Jz * Jz;
    bz[block_id][k] += wv * rv * Jz;

    if (ix == jx) wv = 0;

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1;

    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += wv * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      vi[n] += wv * rv * Ji[n];
      vj[n] += wv * rv * Jj[n];

      Eii[block_id][n][k] += wv * Jz * Ji[n];
      Eij[block_id][n][k] += wv * Jz * Jj[n];
    }
    

    if (opt_intr){
      // calib jacobian pinhole
      if (model_id == 0){
        Jc[0] = -R[1][0] * (u-cx) * fy * d * (1.0/fx) * (1.0/fx) + fy * y * d2 * R[2][0] * (u-cx) * (1.0/fx) * (1.0/fx);
        Jc[1] = -R[1][1] * (v-cy) * (1.0/fy) * d + R[2][1] * (v-cy) * (1.0/fy) * y * d2 + y * d;
        Jc[2] = -R[1][0] * fy * (1.0/fx) * d + fy * y * d2 * R[2][0] * (1.0/fx);
        Jc[3] = -R[1][1] * d + y * d2 * R[2][1] + 1.0;
      }
      else{
        // calib jacobian mei model: numerical approximation because analytical formulas are pages long
        const float mu = 0.0001;
        const float disp = disps[ix][i][j];
        float u_low;
        float u_up;

        proj_transform_mei(uvi, qij, tij, fx-mu, fy, cx,  cy,  xi, disp, uvj);
        u_low = uvj[1];
        proj_transform_mei(uvi, qij, tij, fx+mu, fy, cx,  cy,  xi, disp, uvj);
        u_up = uvj[1];
        Jc[0] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy-mu, cx,  cy,  xi, disp, uvj);
        u_low = uvj[1];
        proj_transform_mei(uvi, qij, tij, fx, fy+mu, cx,  cy,  xi, disp, uvj);
        u_up = uvj[1];
        Jc[1] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy, cx-mu,  cy,  xi, disp, uvj);
        u_low = uvj[1];
        proj_transform_mei(uvi, qij, tij, fx, fy, cx+mu,  cy,  xi, disp, uvj);
        u_up = uvj[1];
        Jc[2] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy-mu,  xi, disp, uvj);
        u_low = uvj[1];
        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy+mu,  xi, disp, uvj);
        u_up = uvj[1];
        Jc[3] = (u_up - u_low) / (2 * mu);

        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy,  xi-0.01*mu, disp, uvj);
        u_low = uvj[1];
        proj_transform_mei(uvi, qij, tij, fx, fy, cx,  cy,  xi+0.01*mu, disp, uvj);
        u_up = uvj[1];
        Jc[4] = (u_up - u_low) / (2 * 0.01* mu);
      }

      l=0;
      for (int n=0; n<n_intr; n++) {
        for (int m=0; m<=n; m++) {
          cij[l] += wv * Jc[n] * Jc[m];
          l++;
        }
      }

      // calib-pose block
      for (int n=0; n<n_intr; n++) {
        for (int m=0; m<6; m++) {
          cpij[0][n][m] += wv * Jc[n] * Ji[m];
          cpij[1][n][m] += wv * Jc[n] * Jj[m];
        }
      }

      for (int n=0; n<n_intr; n++) {
        q[n] += wv * rv * Jc[n];
        CalibDepth[block_id][k][n] += wv * Jz * Jc[n];
      }

    }
  }



  __syncthreads();

  // sum results of all threads and write to output array
  __shared__ float sdata[THREADS];
  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = vi[n];
    // blockReduce sums all entries of sdata efficiently
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[0][block_id][n] = sdata[0];
    }

    __syncthreads();

    sdata[threadIdx.x] = vj[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[1][block_id][n] = sdata[0];
    }

  }

  l=0;
  for (int n=0; n<12; n++) {
    for (int m=0; m<=n; m++) {
      sdata[threadIdx.x] = hij[l];
      blockReduce(sdata);

      if (threadIdx.x == 0) {
        if (n<6 && m<6) {
          Hs[0][block_id][n][m] = sdata[0];
          Hs[0][block_id][m][n] = sdata[0];
        }
        else if (n >=6 && m<6) {
          Hs[1][block_id][m][n-6] = sdata[0];
          Hs[2][block_id][n-6][m] = sdata[0];
        }
        else {
          Hs[3][block_id][n-6][m-6] = sdata[0];
          Hs[3][block_id][m-6][n-6] = sdata[0];
        }
      }

      l++;
    }
  }
  

  if (opt_intr){

    __syncthreads();
    l=0;
    for (int n=0; n<n_intr; n++) {
      for (int m=0; m<=n; m++) {
        sdata[threadIdx.x] = cij[l];
        blockReduce(sdata);

        if (threadIdx.x == 0) {
          Calib[block_id][n][m] = sdata[0];
          Calib[block_id][m][n] = sdata[0];
        }
        l++;
      }
    }

    __syncthreads();
    for (int l = 0; l<2; l++){
      for (int n=0; n<n_intr; n++) {
        for (int m=0; m<6; m++) {
          sdata[threadIdx.x] = cpij[l][n][m];
          blockReduce(sdata);

          if (threadIdx.x == 0) {
            CalibPose[block_id][l][n][m] = sdata[0];
          }
        }
      }
    }

    __syncthreads();

    for (int n=0; n<n_intr; n++) {
      sdata[threadIdx.x] = q[n];
      blockReduce(sdata);
      if (threadIdx.x == 0) {
        qs[block_id][n] = sdata[0];
      }
    }
    __syncthreads();
  }

}


__global__ void projmap_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> valid)
{

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  // load poses from global memory
  if (thread_id < 3) {
    ti[thread_id] = poses[ix][thread_id];
    tj[thread_id] = poses[jx][thread_id];
  }

  if (thread_id < 4) {
    qi[thread_id] = poses[ix][thread_id+3];
    qj[thread_id] = poses[jx][thread_id+3];
  }

  __syncthreads();

  if (thread_id == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {
    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    Xi[0] = (u - cx) / fx;
    Xi[1] = (v - cy) / fy;
    Xi[2] = 1;
    Xi[3] = disps[ix][i][j];

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    coords[block_id][i][j][0] = u;
    coords[block_id][i][j][1] = v;

    if (Xj[2] > 0.01) {
      coords[block_id][i][j][0] = fx * (Xj[0] / Xj[2]) + cx;
      coords[block_id][i][j][1] = fy * (Xj[1] / Xj[2]) + cy;
    }

    valid[block_id][i][j][0] = (Xj[2] > MIN_DEPTH) ? 1.0 : 0.0;

  }
}

__global__ void frame_distance_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dist,
    const float beta,
    const int model_id) {

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;
  __shared__ float xi;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  // FIXME is this thread_id a problem???
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
    if (model_id > 0){
      xi = intrinsics[4];
    }
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  __shared__ float accum[THREADS]; accum[thread_id] = 0;
  __shared__ float valid[THREADS]; valid[thread_id] = 0;
  __shared__ float total[THREADS]; total[thread_id] = 0;

  __syncthreads();

  for (int n=0; n<1; n++) {

    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();


    relSE3(ti, qi, tj, qj, tij, qij);

    float d, du, dv;

    GPU_1D_KERNEL_LOOP(k, ht*wd) {
      const int i = k / wd;
      const int j = k % wd;

      const float u = static_cast<float>(j);
      const float v = static_cast<float>(i);


      
      if (model_id == 0){
        // homogenous coordinates
        Xi[0] = (u - cx) / fx;
        Xi[1] = (v - cy) / fy;
        Xi[2] = 1;
        Xi[3] = disps[ix][i][j];

        // transform homogenous point
        actSE3(tij, qij, Xi, Xj);

        du = fx * (Xj[0] / Xj[2]) + cx - u;
        dv = fy * (Xj[1] / Xj[2]) + cy - v;
        }

      else{
        const float xs = (u - cx) / fx;
        const float ys = (v - cy) / fy;
        const float beta = (xi + sqrt( 1 + ( 1 - xi * xi) * (xs * xs  + ys * ys)) ) / (1 + xs * xs  + ys * ys);

        Xi[0] = beta * xs / (beta - xi);
        Xi[1] = beta * ys / (beta - xi);
        Xi[2] = 1.0;
        Xi[3] = disps[ix][i][j];

        // transform homogenous point
        actSE3(tij, qij, Xi, Xj);

        const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
        const float z = (Xj[2] < MIN_DEPTH) ? 0.0 : Xj[2];
        const float r = sqrt(Xj[0] * Xj[0] + Xj[1] * Xj[1] + z * z);
        const float mei_fact = ((z + xi * r) < MIN_DEPTH) ? 0.0 : 1.0 / (z + xi * r);

        du = fx * Xj[0] * mei_fact + cx - u;
        dv = fy * Xj[1] * mei_fact + cy - v;
      }

      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += beta;
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += beta * d;
        valid[threadIdx.x] += beta;
      }

      if (model_id == 0){
        Xi[0] = (u - cx) / fx;
        Xi[1] = (v - cy) / fy;
        Xi[2] = 1;
        Xi[3] = disps[ix][i][j];

        Xj[0] = Xi[0] + Xi[3] * tij[0];
        Xj[1] = Xi[1] + Xi[3] * tij[1];
        Xj[2] = Xi[2] + Xi[3] * tij[2];

        du = fx * (Xj[0] / Xj[2]) + cx - u;
        dv = fy * (Xj[1] / Xj[2]) + cy - v;
        }
      else{
        const float xs = (u - cx) / fx;
        const float ys = (v - cy) / fy;
        const float beta = (xi + sqrt( 1 + ( 1 - xi * xi) * (xs * xs  + ys * ys)) ) / (1 + xs * xs  + ys * ys);

        Xi[0] = beta * xs / (beta - xi);
        Xi[1] = beta * ys / (beta - xi);
        Xi[2] = 1.0;
        Xi[3] = disps[ix][i][j];

        Xj[0] = Xi[0] + Xi[3] * tij[0];
        Xj[1] = Xi[1] + Xi[3] * tij[1];
        Xj[2] = Xi[2] + Xi[3] * tij[2];

        const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
        const float z = (Xj[2] < MIN_DEPTH) ? 0.0 : Xj[2];
        const float r = sqrt(Xj[0] * Xj[0] + Xj[1] * Xj[1] + z * z);
        const float mei_fact = ((z + xi * r) < MIN_DEPTH) ? 0.0 : 1.0 / (z + xi * r);

        du = fx * Xj[0] * mei_fact + cx - u;
        dv = fy * Xj[1] * mei_fact + cy - v;
        }

      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += (1 - beta);
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += (1 - beta) * d;
        valid[threadIdx.x] += (1 - beta);
      }
    }

    if (threadIdx.x == 0) {
      int tmp = ix;
      ix = jx;
      jx = tmp;
    }

    __syncthreads();

  }
  __syncthreads(); blockReduce(accum);
  __syncthreads(); blockReduce(total);
  __syncthreads(); blockReduce(valid);

  __syncthreads();

  if (thread_id == 0) {
    dist[block_id] = (valid[0] / (total[0] + 1e-8) < 0.75) ? 1000.0 : accum[0] / valid[0];
  }
}



__global__ void depth_filter_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> thresh,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> counter)
{

  const int block_id = blockIdx.x;
  const int neigh_id = blockIdx.y;
  const int index = blockIdx.z * blockDim.x + threadIdx.x;

  // if (threadIdx.x == 0) {
  //   printf("%d %d %d %d\n", blockIdx.x, blockIdx.y, blockDim.x, threadIdx.x);
  // }

  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  if (threadIdx.x == 0) {
    ix = static_cast<int>(inds[block_id]);
    jx = (neigh_id < 3) ? ix - neigh_id - 1 : ix + neigh_id;
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  if (jx < 0 || jx >= num) {
    return;
  }

  const float t = thresh[block_id];

  // load poses from global memory
  if (threadIdx.x < 3) {
    ti[threadIdx.x] = poses[ix][threadIdx.x];
    tj[threadIdx.x] = poses[jx][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    qi[threadIdx.x] = poses[ix][threadIdx.x+3];
    qj[threadIdx.x] = poses[jx][threadIdx.x+3];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];

  __syncthreads();

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[ix][i][j];
    
    // homogenous coordinates
    Xi[0] = (ui - cx) / fx;
    Xi[1] = (vi - cy) / fy;
    Xi[2] = 1;
    Xi[3] = di;

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    const float uj = fx * (Xj[0] / Xj[2]) + cx;
    const float vj = fy * (Xj[1] / Xj[2]) + cy;
    const float dj = Xj[3] / Xj[2];

    const int u0 = static_cast<int>(floor(uj));
    const int v0 = static_cast<int>(floor(vj));

    if (u0 >= 0 && v0 >= 0 && u0 < wd-1 && v0 < ht-1) {
      const float wx = ceil(uj) - uj;
      const float wy = ceil(vj) - vj;

      const float d00 = disps[jx][v0+0][u0+0];
      const float d01 = disps[jx][v0+0][u0+1];
      const float d10 = disps[jx][v0+1][u0+0];
      const float d11 = disps[jx][v0+1][u0+1];

      const float dj_hat = wy*wx*d00 + wy*(1-wx)*d01 + (1-wy)*wx*d10 + (1-wy)*(1-wx)*d11;

      const float err = abs(1.0/dj - 1.0/dj_hat);
      if       (abs(1.0/dj - 1.0/d00) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d01) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d10) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d11) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
    }
  }
}



__global__ void iproj_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> points)

{

  const int block_id = blockIdx.x;
  const int index = blockIdx.y * blockDim.x + threadIdx.x;


  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float t[3];
  __shared__ float q[4];

  if (threadIdx.x == 0) {
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();


  // load poses from global memory
  if (threadIdx.x < 3) {
    t[threadIdx.x] = poses[block_id][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    q[threadIdx.x] = poses[block_id][threadIdx.x+3];
  }

  __syncthreads();

  // points 
  float Xi[4];
  float Xj[4];

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[block_id][i][j];
    
    // homogenous coordinates
    Xi[0] = (ui - cx) / fx;
    Xi[1] = (vi - cy) / fy;
    Xi[2] = 1;
    Xi[3] = di;

    // transform homogenous point
    actSE3(t, q, Xi, Xj);

    points[block_id][i][j][0] = Xj[0] / Xj[3];
    points[block_id][i][j][1] = Xj[1] / Xj[3];
    points[block_id][i][j][2] = Xj[2] / Xj[3];

  }
}



__global__ void accum_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> inps,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ptrs,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idxs,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> outs)
{
  
  const int block_id = blockIdx.x;
  const int D = inps.size(2);

  const int start = ptrs[block_id];
  const int end = ptrs[block_id+1];

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float x = 0;
    for (int i=start; i<end; i++) {
      x += inps[idxs[i]][k];
    }
    outs[block_id][k] = x;
  }  
}


__device__ void
retrSE3(const float *xi, const float* t, const float* q, float* t1, float* q1) {
  // retraction on SE3 manifold

  float dt[3] = {0, 0, 0};
  float dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}


__global__ void pose_retr_kernel(
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dx,
    const int t0, const int t1) 
{

  for (int k=t0+threadIdx.x; k<t1; k+=blockDim.x) {
    float xi[6], q[4], q1[4], t[3], t1[3];

    t[0] = poses[k][0];
    t[1] = poses[k][1];
    t[2] = poses[k][2];

    q[0] = poses[k][3];
    q[1] = poses[k][4];
    q[2] = poses[k][5];
    q[3] = poses[k][6];
    
    for (int n=0; n<6; n++) {
      xi[n] = dx[k-t0][n];
    }

    retrSE3(xi, t, q, t1, q1);

    poses[k][0] = t1[0];
    poses[k][1] = t1[1];
    poses[k][2] = t1[2];

    poses[k][3] = q1[0];
    poses[k][4] = q1[1];
    poses[k][5] = q1[2];
    poses[k][6] = q1[3];
  }
}

__global__ void disp_retr_kernel(
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dz,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds) 
{
  const int i = inds[blockIdx.x];
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  for (int k=threadIdx.x; k<ht*wd; k+=blockDim.x) {
    float d = disps[i][k/wd][k%wd] + dz[blockIdx.x][k];
    disps[i][k/wd][k%wd] = d;
  }
}

torch::Tensor accum_cuda(torch::Tensor data, torch::Tensor ix, torch::Tensor jx) {
  torch::Tensor ix_cpu = ix.to(torch::kCPU);
  torch::Tensor jx_cpu = jx.to(torch::kCPU);
  torch::Tensor inds = torch::argsort(ix_cpu);

  long* ix_data = ix_cpu.data_ptr<long>();
  long* jx_data = jx_cpu.data_ptr<long>();
  long* kx_data = inds.data_ptr<long>();

  int count = jx.size(0);
  std::vector<int> cols;

  torch::Tensor ptrs_cpu = torch::zeros({count+1}, 
    torch::TensorOptions().dtype(torch::kInt64));
  
  long* ptrs_data = ptrs_cpu.data_ptr<long>();
  ptrs_data[0] = 0;

  int i = 0;
  for (int j=0; j<count; j++) {
    while (i < ix.size(0) && ix_data[kx_data[i]] <= jx_data[j]) {
      if (ix_data[kx_data[i]] == jx_data[j])
        cols.push_back(kx_data[i]);
      i++;
    }
    ptrs_data[j+1] = cols.size();
  }

  torch::Tensor idxs_cpu = torch::zeros({long(cols.size())}, 
    torch::TensorOptions().dtype(torch::kInt64));

  long* idxs_data = idxs_cpu.data_ptr<long>();

  for (int i=0; i<cols.size(); i++) {
    idxs_data[i] = cols[i];
  }

  torch::Tensor ptrs = ptrs_cpu.to(torch::kCUDA);
  torch::Tensor idxs = idxs_cpu.to(torch::kCUDA);

  torch::Tensor out = torch::zeros({jx.size(0), data.size(1)},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  accum_kernel<<<count, THREADS>>>(
    data.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ptrs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    idxs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    out.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  return out;
}


__global__ void EEt6x6_kernel(
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> S)
{

  // indicies
  const int ix = idx[blockIdx.x][0];
  const int jx = idx[blockIdx.x][1];
  const int kx = idx[blockIdx.x][2];

  const int D = E.size(2);

  float dS[6][6];
  float ei[6];
  float ej[6];

  for (int i=0; i<6; i++) {
    for (int j=0; j<6; j++) {
      dS[i][j] = 0;
    }
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q = Q[kx][k];
      
    // coalesced memory read
    for (int n=0; n<6; n++) {
      ei[n] = E[ix][n][k] * q;
      ej[n] = E[jx][n][k];
    }

    // block EEt
    for (int n=0; n<6; n++) {
      for (int m=0; m<6; m++) {
        dS[n][m] += ei[n] * ej[m];
      }
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    for (int m=0; m<6; m++) {
      sdata[threadIdx.x] = dS[n][m];

      blockReduce(sdata);

      if (threadIdx.x == 0) {
        S[blockIdx.x][n][m] = sdata[0];
      }
    }
  }
}

__global__ void  ECDT_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float, 3,torch::RestrictPtrTraits> CD, 
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> ECDT_ext)
  {
  const int D = E.size(2); 
  const int n_intr = CD.size(1);
  const int kx = idx[blockIdx.x][0];

  float b[6][5]; 

  for (int n=0; n<6; n++) {
    for (int m=0; m<n_intr; m++) {
      b[n][m] = 0.0;
    }
  }


  for (int k=threadIdx.x; k<D; k+=blockDim.x) {

    for (int m=0; m<n_intr; m++) {
      const float q_w = Q[kx][k] * CD[kx][m][k];

      for (int n=0; n<6; n++) {
        b[n][m] += q_w * E[blockIdx.x][n][k];
      }
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    for (int m=0; m<n_intr; m++) {

      sdata[threadIdx.x] = b[n][m];
      blockReduce(sdata);

      if (threadIdx.x == 0) {
        ECDT_ext[blockIdx.x][n][m] += sdata[0];
      }
    }
  }
}



__global__ void Ev6x1_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float, 2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> v)
{
  const int D = E.size(2);
  const int kx = idx[blockIdx.x][0];

  float b[6];
  for (int n=0; n<6; n++) {
    b[n] = 0.0;
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q_w = Q[kx][k] * w[kx][k];

    for (int n=0; n<6; n++) {
      b[n] += q_w * E[blockIdx.x][n][k];
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = b[n];
    blockReduce(sdata);

    if (threadIdx.x == 0) {
      v[blockIdx.x][n] += sdata[0];
    }
  }
}

__global__ void EvT6x1_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E, 
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> x, 
  const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idx, 
  torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w) 
{

  const int D = E.size(2);
  const int ix = idx[blockIdx.x];

  if (idx[blockIdx.x] <= 0 || idx[blockIdx.x] >= x.size(0))
    return;

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float dw = 0;
    for (int n=0; n<6; n++) {
      dw += E[blockIdx.x][n][k] * x[ix][n];
    }
    w[blockIdx.x][k] = dw;
  }
}

__global__ void CDCDT_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> CD, 
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q, 
  torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> CDCDT) 
{
  const int D = CD.size(2); 
  const int n_intr = CD.size(1);
  float v_thread[5][5]; 


  for (int n=0; n<n_intr; n++) {
    for (int m=0; m<=n; m++) {
      float val = 0;

      for (int k=threadIdx.x; k<D; k+=blockDim.x) {
        val += CD[blockIdx.x][n][k] * Q[blockIdx.x][k] * CD[blockIdx.x][m][k];
      }
      v_thread[n][m] = val;
      v_thread[m][n] = val;
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<n_intr; n++) {
    for (int m=0; m<=n; m++) {
      sdata[threadIdx.x] = v_thread[n][m];
      blockReduce(sdata);

      if (threadIdx.x == 0) {
        CDCDT[blockIdx.x][n][m] = sdata[0];
        CDCDT[blockIdx.x][m][n] = sdata[0];
      }
    }
  }

}

__global__ void CDv_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> CD, 
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q, 
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w, 
  torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> v_ext) 

{
  const int D = CD.size(2); 
  const int n_intr = CD.size(1);
  float v_thread[5];

  for (int n=0; n<n_intr; n++) {
    float v_exti = 0;

    for (int k=threadIdx.x; k<D; k+=blockDim.x) {
      v_exti += CD[blockIdx.x][n][k] * Q[blockIdx.x][k] * w[blockIdx.x][k];
    }
    v_thread[n] = v_exti;
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<n_intr; n++) {
    sdata[threadIdx.x] = v_thread[n];
    blockReduce(sdata);

    if (threadIdx.x == 0) {
      v_ext[blockIdx.x][n] += sdata[0];
    }
  }


}

__global__ void CDvT_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> CD, 
  const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dI, 
  torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dw_add) 
{

  const int D = CD.size(2); 
  const int n_intr = CD.size(1);

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float dw = 0;
    for (int n=0; n<n_intr; n++) {
      dw += CD[blockIdx.x][n][k] * dI[n];
    }
    dw_add[blockIdx.x][k] = dw;
  }
  __syncthreads();
}


torch::Tensor accum_CalibPose(torch::Tensor CPii, torch::Tensor ii, const int n_unique) {
  
  const int n_intr = CPii.size(1);

  auto CPii_cpu = CPii.to(torch::kCPU).to(torch::kFloat64);
  auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

  auto CPii_acc = CPii_cpu.accessor<double,3>();
  auto ii_acc = ii_cpu.accessor<long,1>();

  torch::Tensor CP = torch::zeros({n_unique*6, n_intr});
  
  for (int p=0; p<n_intr; p++) {

    for (int n=0; n<ii.size(0); n++) {
      const int i = ii_acc[n];
      if (i >= 0) {
        for (int j=0; j<6; j++) {
          CP[i*6 + j][p] += CPii_acc[n][p][j];
        }
      }
    }
  }

  return CP;
}



class SparseBlock {
  public:

    Eigen::SparseMatrix<double> A;
    Eigen::VectorX<double> b;

    SparseBlock(int N, int M) : N(N), M(M) {
      A = Eigen::SparseMatrix<double>(N*M, N*M);
      b = Eigen::VectorXd::Zero(N*M);
    }

    SparseBlock(Eigen::SparseMatrix<double> const& A, Eigen::VectorX<double> const& b, 
        int N, int M) : A(A), b(b), N(N), M(M) {}

    void update_lhs(torch::Tensor As, torch::Tensor ii, torch::Tensor jj) {

      auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

      auto As_acc = As_cpu.accessor<double,3>();
      auto ii_acc = ii_cpu.accessor<long,1>();
      auto jj_acc = jj_cpu.accessor<long,1>();

      std::vector<T> tripletList;
      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        const int j = jj_acc[n];

        if (i >= 0 && j >= 0) {
          for (int k=0; k<M; k++) {
            for (int l=0; l<M; l++) {
              double val = As_acc[n][k][l];
              tripletList.push_back(T(M*i + k, M*j + l, val));
            }
          }
        }
      }
      A.setFromTriplets(tripletList.begin(), tripletList.end());
    }

    void update_rhs(torch::Tensor bs, torch::Tensor ii) {
      auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

      auto bs_acc = bs_cpu.accessor<double,2>();
      auto ii_acc = ii_cpu.accessor<long,1>();

      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        if (i >= 0) {
          for (int j=0; j<M; j++) {
            b(i*M + j) += bs_acc[n][j];
          }
        }
      }
    }

    SparseBlock operator-(const SparseBlock& S) {
      return SparseBlock(A - S.A, b - S.b, N, M);
    }

    std::tuple<torch::Tensor, torch::Tensor> get_dense() {
      Eigen::MatrixXd Ad = Eigen::MatrixXd(A);

      torch::Tensor H = torch::from_blob(Ad.data(), {N*M, N*M}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      torch::Tensor v = torch::from_blob(b.data(), {N*M, 1}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      return std::make_tuple(H, v);

    }

    torch::Tensor solve(const float lm=0.0001, const float ep=0.1) {

      torch::Tensor dx;

      Eigen::SparseMatrix<double> L(A);
      // FIXME: Eigen does not support easy interaction between SparseMatrix and Diagonal
      // see https://gitlab.com/libeigen/eigen/-/issues/1574
      // L.diagonal().array() += ep + lm * L.diagonal().array();
      for (int i = 0; i < std::min(L.rows(), L.cols()); ++i) {
        L.coeffRef(i, i) += ep + lm * L.coeffRef(i, i);
      }

      Eigen::SimplicialLLT<Eigen::SparseMatrix<double>> solver;
      solver.compute(L); // Compute the sparse Cholesky decomposition of matrix L

      if (solver.info() == Eigen::Success) {
        Eigen::VectorXd x = solver.solve(b);
        dx = torch::from_blob(x.data(), {N, M}, torch::TensorOptions()
          .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);
      }
      else {
        std::cout << "Warning: Cholesky decomposition was not successful." << std::endl;
        dx = torch::zeros({N, M}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));
      }
      
      return dx;
    }

  private:
    const int N;
    const int M;

};



class SparseBlockAsym {
  public:

    Eigen::SparseMatrix<double> A;
    Eigen::VectorX<double> b;

    SparseBlockAsym(int N) : N(N) {
      A = Eigen::SparseMatrix<double>(N, N);
      b = Eigen::VectorXd::Zero(N);
    }

    SparseBlockAsym(Eigen::SparseMatrix<double> const& A, Eigen::VectorX<double> const& b, 
        int N) : A(A), b(b), N(N){}

    void update_lhs_sym(torch::Tensor As, torch::Tensor ii, torch::Tensor jj, const int M) {

      auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

      auto As_acc = As_cpu.accessor<double,3>();
      auto ii_acc = ii_cpu.accessor<long,1>();
      auto jj_acc = jj_cpu.accessor<long,1>();

      std::vector<T> tripletList;
      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        const int j = jj_acc[n];

        if (i >= 0 && j >= 0) {
          for (int k=0; k<M; k++) {
            for (int l=0; l<M; l++) {
              double val = As_acc[n][k][l];
              tripletList.push_back(T(M*i + k, M*j + l, val));
            }
          }
        }
      }
      A.setFromTriplets(tripletList.begin(), tripletList.end());
    }

    void update_lhs_extended(torch::Tensor As, torch::Tensor ii, torch::Tensor jj, 
                             const int M, std::vector<T> extTripletList) 
    {

      auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

      auto As_acc = As_cpu.accessor<double,3>();
      auto ii_acc = ii_cpu.accessor<long,1>();
      auto jj_acc = jj_cpu.accessor<long,1>();

      std::vector<T> tripletList;
      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        const int j = jj_acc[n];

        if (i >= 0 && j >= 0) {
          for (int k=0; k<M; k++) {
            for (int l=0; l<M; l++) {
              double val = As_acc[n][k][l];
              tripletList.push_back(T(M*i + k, M*j + l, val));
            }
          }
        }
      }
      

      for (int iext=0; iext < extTripletList.size(); iext++){
        tripletList.push_back(extTripletList[iext]);
      }

      A.setFromTriplets(tripletList.begin(), tripletList.end());
    }

    void update_rhs_sym(torch::Tensor bs, torch::Tensor ii, const int M) {
      auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

      auto bs_acc = bs_cpu.accessor<double,2>();
      auto ii_acc = ii_cpu.accessor<long,1>();

      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        if (i >= 0) {
          for (int j=0; j<M; j++) {
            b(i*M + j) += bs_acc[n][j];
          }
        }
      }
    }

    void update_rhs_extended(torch::Tensor bs, torch::Tensor ii, 
                             const int M, torch::Tensor ext_vec) {

      auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto ext_cpu = ext_vec.to(torch::kCPU).to(torch::kFloat64);

      auto bs_acc = bs_cpu.accessor<double,2>();
      auto ii_acc = ii_cpu.accessor<long,1>();
      auto ext_acc = ext_cpu.accessor<double,1>();

      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        if (i >= 0) {
          for (int j=0; j<M; j++) {
            b(i*M + j) += bs_acc[n][j];
          }
        }
      }
      
      for (int n=0; n<ext_acc.size(0); n++){
        b(N - ext_acc.size(0) + n) = ext_acc[n];
      }


    }
    SparseBlockAsym operator-(const SparseBlockAsym& S) {
      return SparseBlockAsym(A - S.A, b - S.b, N);
    }


    torch::Tensor solve(const float lm=0.0001, const float ep=0.1) {

      torch::Tensor dx;

      Eigen::SparseMatrix<double> L(A);
      // FIXME: Eigen does not support easy interaction between SparseMatrix and Diagonal
      // see https://gitlab.com/libeigen/eigen/-/issues/1574
      // L.diagonal().array() += ep + lm * L.diagonal().array();
      for (int i = 0; i < std::min(L.rows(), L.cols()); ++i) {
        L.coeffRef(i, i) += ep + lm * L.coeffRef(i, i);
      }

      Eigen::SimplicialLLT<Eigen::SparseMatrix<double>> solver;
      solver.compute(L);

      if (solver.info() == Eigen::Success) {
        Eigen::VectorXd x = solver.solve(b);
        dx = torch::from_blob(x.data(), {N}, torch::TensorOptions()
          .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);
      }
      else {
        std::cout << "Warning: Cholesky decomposition was not successful." << std::endl;
        dx = torch::zeros({N}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));
      }
      
      return dx;
    }

  private:
    const int N;

};


SparseBlockAsym pose_calib_block(torch::Tensor Hs,
                        torch::Tensor vs,
                        torch::Tensor Calib,
                        torch::Tensor q,
                        torch::Tensor CalibPose,
                        torch::Tensor ii,
                        torch::Tensor jj,
                        const int t0,
                        const int t1){
  

  torch::Tensor C_block = torch::sum(Calib, 0); 
  const int n_intr = C_block.size(0);
  torch::Tensor q_vec = torch::sum(q, 0); 
  int n_unique = t1-t0; 

  torch::Tensor CPii = CalibPose.permute({1, 0, 2, 3}).reshape({-1, n_intr, 6}); 
  torch::Tensor CP = accum_CalibPose(CPii, torch::cat({ii, jj}) - t0, n_unique); 

  // get triplet list to extend pose hessian with calib blocks
  int i0 = n_unique * 6;
  std::vector<T> extTripletList;

  // calib block
  for (int i=0; i<n_intr; i++){
    for (int j=0; j<n_intr; j++){
      double val = C_block[i][j].item<double>();
      extTripletList.push_back(T(i0+i, i0+j, val));
    }
  }

  // calib-pose block
  for (int i=0; i<n_intr; i++){
    for (int j=0; j<i0; j++){
      double val = CP[j][i].item<double>();
      extTripletList.push_back(T(i0+i, j, val));
      extTripletList.push_back(T(j, i0+i, val));
    }
  }

  // pose-calib block
  SparseBlockAsym A(n_unique * 6 + n_intr);
  int M = 6;

  A.update_lhs_extended(Hs.reshape({-1, 6, 6}), 
      torch::cat({ii, ii, jj, jj}) - t0, 
      torch::cat({ii, jj, ii, jj}) - t0,
      M,
      extTripletList);

  A.update_rhs_extended(vs.reshape({-1, 6}), 
      torch::cat({ii, jj}) - t0, M, q_vec);

  return A;
}



SparseBlockAsym schur_calib_block(torch::Tensor E, 
                        torch::Tensor Q, 
                        torch::Tensor CD, 
                        torch::Tensor w, 
                        torch::Tensor ii, 
                        torch::Tensor jj, 
                        torch::Tensor kk, 
                        torch::Tensor kx, 
                        const int t0,
                        const int t1)
{

  torch::Tensor ii_cpu = ii.to(torch::kCPU);
  torch::Tensor jj_cpu = jj.to(torch::kCPU);
  torch::Tensor kk_cpu = kk.to(torch::kCPU);

  const int P = t1 - t0;
  const long* ii_data = ii_cpu.data_ptr<long>();
  const long* jj_data = jj_cpu.data_ptr<long>();
  const long* kk_data = kk_cpu.data_ptr<long>();
  const int ht_wd = E.size(2);
  const int n_intr = CD.size(1);

  std::vector<std::vector<long>> graph(P);
  std::vector<std::vector<long>> index(P);

  for (int n=0; n<ii_cpu.size(0); n++) {
    const int j = jj_data[n];
    const int k = kk_data[n];

    if (j >= t0 && j <= t1) {
      const int t = j - t0;
      graph[t].push_back(k);
      index[t].push_back(n);
    }
  }

  std::vector<long> ii_list, jj_list, idx, jdx;

  for (int i=0; i<P; i++) {
    for (int j=0; j<P; j++) {
      for (int k=0; k < graph[i].size(); k++) {
        for (int l=0; l < graph[j].size(); l++) {
          if (graph[i][k] == graph[j][l]) {
            ii_list.push_back(i);
            jj_list.push_back(j);

            idx.push_back(index[i][k]);
            idx.push_back(index[j][l]);
            idx.push_back(graph[i][k]);
          }
        }
      }
    }
  }

  torch::Tensor ix_cuda = torch::from_blob(idx.data(), {long(idx.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).to(torch::kCUDA).view({-1, 3});

  torch::Tensor jx_cuda = torch::stack({kk_cpu}, -1)
    .to(torch::kCUDA).to(torch::kInt64);

  torch::Tensor ii2_cpu = torch::from_blob(ii_list.data(), {long(ii_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor jj2_cpu = torch::from_blob(jj_list.data(), {long(jj_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor S = torch::zeros({ix_cuda.size(0), 6, 6}, 
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  torch::Tensor v = torch::zeros({jx_cuda.size(0), 6},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));


  // parallel computation of E Q E^T
  EEt6x6_kernel<<<ix_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ix_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    S.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  // parallel computation of E Q w
  Ev6x1_kernel<<<jx_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    jx_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    v.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  // schur block
  SparseBlockAsym A(P * 6 + n_intr);
  int M = 6;

  std::vector<T> extTripletList;

  // 4x4 lower right square of E Q E^T
  torch::Tensor CDCDT = torch::zeros({kx.size(0), n_intr, n_intr}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));

  CDCDT_kernel<<<kx.size(0), THREADS>>>(
    CD.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    CDCDT.packed_accessor32<float,3,torch::RestrictPtrTraits>()
  );
  torch::Tensor CDCDT_flat = torch::sum(CDCDT, 0).to(torch::kCUDA).to(torch::kFloat32);

  for (int i=0; i<n_intr; i++){
    for (int j=0; j<n_intr; j++){
      double val = CDCDT_flat[i][j].item<double>();
      extTripletList.push_back(T(6*P+i, 6*P+j, val));
    }
  }

  torch::Tensor ECDT_ext = torch::zeros({jx_cuda.size(0), 6, n_intr}, torch::TensorOptions()
         .device(torch::kCUDA).dtype(torch::kFloat32));

  ECDT_kernel<<<jx_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    CD.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    jx_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    ECDT_ext.packed_accessor32<float,3,torch::RestrictPtrTraits>()
  );


  torch::Tensor ECDT = torch::zeros({6*P, n_intr}, torch::TensorOptions()
         .device(torch::kCUDA).dtype(torch::kFloat32));

  for (int n=0; n<n_intr; n++){
    for (int i=0; i<jx_cuda.size(0); i++){
      int ki = jj_cpu[i].item<int>() - t0; 
      if (ki >= 0){
        for (int l=0; l<6; l++){
          ECDT[ki*6+l][n] += ECDT_ext[i][l][n];
        }
      }
    }
  }
  

  for (int j=0; j<n_intr; j++){
    for (int i=0; i<6*P; i++){
      double val = ECDT[i][j].item<double>();
      extTripletList.push_back(T(i, 6*P+j, val));
      extTripletList.push_back(T(6*P+j, i, val));
    }
  }

  torch::Tensor v_ext = torch::zeros({kx.size(0), n_intr}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));

  CDv_kernel<<<kx.size(0), THREADS>>>(
    CD.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    v_ext.packed_accessor32<float,2,torch::RestrictPtrTraits>());
  
  torch::Tensor v_ext_flat = torch::sum(v_ext, 0).to(torch::kCUDA).to(torch::kFloat32);

  A.update_lhs_extended(S, ii2_cpu, jj2_cpu, M, extTripletList);
  A.update_rhs_extended(v, jj_cpu - t0, M, v_ext_flat);

  return A;

}


SparseBlock schur_block(torch::Tensor E,
                        torch::Tensor Q,
                        torch::Tensor w,
                        torch::Tensor ii,
                        torch::Tensor jj,
                        torch::Tensor kk,
                        const int t0,
                        const int t1)
{

  torch::Tensor ii_cpu = ii.to(torch::kCPU);
  torch::Tensor jj_cpu = jj.to(torch::kCPU);
  torch::Tensor kk_cpu = kk.to(torch::kCPU);

  const int P = t1 - t0;
  const long* ii_data = ii_cpu.data_ptr<long>();
  const long* jj_data = jj_cpu.data_ptr<long>();
  const long* kk_data = kk_cpu.data_ptr<long>();

  std::vector<std::vector<long>> graph(P);
  std::vector<std::vector<long>> index(P);

  for (int n=0; n<ii_cpu.size(0); n++) {
    const int j = jj_data[n];
    const int k = kk_data[n];

    if (j >= t0 && j <= t1) {
      const int t = j - t0;
      graph[t].push_back(k);
      index[t].push_back(n);
    }
  }

  std::vector<long> ii_list, jj_list, idx, jdx;

  for (int i=0; i<P; i++) {
    for (int j=0; j<P; j++) {
      for (int k=0; k < graph[i].size(); k++) {
        for (int l=0; l < graph[j].size(); l++) {
          if (graph[i][k] == graph[j][l]) {
            ii_list.push_back(i);
            jj_list.push_back(j);

            idx.push_back(index[i][k]);
            idx.push_back(index[j][l]);
            idx.push_back(graph[i][k]);
          }
        }
      }
    }
  }

  torch::Tensor ix_cuda = torch::from_blob(idx.data(), {long(idx.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).to(torch::kCUDA).view({-1, 3});

  torch::Tensor jx_cuda = torch::stack({kk_cpu}, -1)
    .to(torch::kCUDA).to(torch::kInt64);

  torch::Tensor ii2_cpu = torch::from_blob(ii_list.data(), {long(ii_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor jj2_cpu = torch::from_blob(jj_list.data(), {long(jj_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor S = torch::zeros({ix_cuda.size(0), 6, 6}, 
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  torch::Tensor v = torch::zeros({jx_cuda.size(0), 6},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));


  EEt6x6_kernel<<<ix_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ix_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    S.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  Ev6x1_kernel<<<jx_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    jx_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    v.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  SparseBlock A(P, 6);

  A.update_lhs(S, ii2_cpu, jj2_cpu);
  A.update_rhs(v, jj_cpu - t0);

  return A;
}


std::vector<torch::Tensor> ba_cuda(
    torch::Tensor poses, 
    torch::Tensor disps, 
    torch::Tensor intrinsics, 
    torch::Tensor disps_sens, 
    torch::Tensor targets, 
    torch::Tensor weights, 
    torch::Tensor eta,
    torch::Tensor ii, 
    torch::Tensor jj, 
    const int t0,
    const int t1,
    const int iterations,
    const int model_id,
    const float lm,
    const float ep,
    const bool motion_only,
    const bool opt_intr)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);
  const int n_intr = intrinsics.size(0);

  torch::Tensor ts = torch::arange(t0, t1).to(torch::kCUDA);
  torch::Tensor ii_exp = torch::cat({ts, ii}, 0);
  torch::Tensor jj_exp = torch::cat({ts, jj}, 0);

  std::tuple<torch::Tensor, torch::Tensor> kuniq = 
    torch::_unique(ii_exp, true, true);

  torch::Tensor kx = std::get<0>(kuniq); 
  torch::Tensor kk_exp = std::get<1>(kuniq); 
    
  torch::Tensor dx;
  torch::Tensor dz;
  torch::Tensor dI;
  torch::Tensor dxdI;

  // initialize buffers
  torch::Tensor Hs = torch::zeros({4, num, 6, 6}, opts);
  torch::Tensor vs = torch::zeros({2, num, 6}, opts);
  torch::Tensor Eii = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Eij = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Cii = torch::zeros({num, ht*wd}, opts);
  torch::Tensor wi = torch::zeros({num, ht*wd}, opts);

  torch::Tensor Calib = torch::zeros({num, n_intr, n_intr}, opts);
  torch::Tensor CalibPose = torch::zeros({num, 2, n_intr, 6}, opts);
  torch::Tensor CalibDepth = torch::zeros({num, ht*wd, n_intr}, opts);
  torch::Tensor q = torch::zeros({num, n_intr}, opts);

  for (int itr=0; itr<iterations; itr++) {

    projective_transform_kernel<<<num, THREADS>>>(
      targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      Hs.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      vs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eii.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eij.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      wi.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      Calib.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      CalibPose.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      CalibDepth.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      opt_intr,
      n_intr,
      model_id
      );

    SparseBlock A(t1 - t0, 6);

    A.update_lhs(Hs.reshape({-1, 6, 6}), 
        torch::cat({ii, ii, jj, jj}) - t0, 
        torch::cat({ii, jj, ii, jj}) - t0);

    A.update_rhs(vs.reshape({-1, 6}), 
        torch::cat({ii, jj}) - t0);

    if (motion_only) {
      dx = A.solve(lm, ep);

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);
    }

    else if (opt_intr) {
      const float alpha = 0.05;
      torch::Tensor m = (disps_sens.index({kx, "..."}) > 0).to(torch::TensorOptions().dtype(\
                        torch::kFloat32)).view({-1, ht*wd}); 

      torch::Tensor C = accum_cuda(Cii, ii, kx) + m * alpha + (1 - m) * eta.view({-1, ht*wd}); 

      torch::Tensor w = accum_cuda(wi, ii, kx) - m * alpha * (disps.index({kx, "..."}) \
                        - disps_sens.index({kx, "..."})).view({-1, ht*wd}); 

      torch::Tensor Q = 1.0 / C;

      torch::Tensor Ei = accum_cuda(Eii.view({num, 6*ht*wd}), ii, ts).view({t1-t0, 6, ht*wd});
      torch::Tensor E = torch::cat({Ei, Eij}, 0);

      torch::Tensor CD_block = accum_cuda(CalibDepth.view({num, n_intr*ht*wd}), ii, kx).view(\
                               {-1, ht*wd, n_intr}).permute({0, 2, 1});

      SparseBlockAsym A = pose_calib_block(Hs, vs, Calib, q, CalibPose, ii, jj, t0, t1);
      SparseBlockAsym S = schur_calib_block(E, Q, CD_block, w, ii_exp, jj_exp, kk_exp, kx, t0, t1);

      dxdI = (A - S).solve(lm, ep);

      dx = torch::zeros({(t1 - t0)*6}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));

      dI = torch::zeros({n_intr}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));
      
      for (int i=0; i<(t1 - t0)*6; i++){
        dx[i] = dxdI[i];
      }

      dx = dx.view({(t1 - t0), 6});

      for (int i=0; i<n_intr; i++){
        dI[i] = dxdI[(t1 - t0)*6+i];
      }
      
      torch::Tensor ix = jj_exp - t0;
      torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

      EvT6x1_kernel<<<ix.size(0), THREADS>>>(
        E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());
      
      torch::Tensor dw_add = torch::zeros({kx.size(0), ht*wd}, opts);

      CDvT_kernel<<<kx.size(0), THREADS>>>(
        CD_block.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dI.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
        dw_add.packed_accessor32<float,2,torch::RestrictPtrTraits>());
      

      dz = Q * (w - accum_cuda(dw, ii_exp, kx) + dw_add); 

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);

      // update disparity maps
      disp_retr_kernel<<<kx.size(0), THREADS>>>(
        disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        kx.packed_accessor32<long,1,torch::RestrictPtrTraits>());
      
      // update intrinsics
      for (int i=0; i<n_intr; i++){
        intrinsics[i] += dI[i];
      }

      
    }
    
    else {
      const float alpha = 0.05;
      torch::Tensor m = (disps_sens.index({kx, "..."}) > 0).to(torch::TensorOptions().dtype(\
                        torch::kFloat32)).view({-1, ht*wd});
      torch::Tensor C = accum_cuda(Cii, ii, kx) + m * alpha + (1 - m) * eta.view({-1, ht*wd});
      torch::Tensor w = accum_cuda(wi, ii, kx) - m * alpha * (disps.index({kx, "..."}) - \
                        disps_sens.index({kx, "..."})).view({-1, ht*wd});
      torch::Tensor Q = 1.0 / C;

      torch::Tensor Ei = accum_cuda(Eii.view({num, 6*ht*wd}), ii, ts).view({t1-t0, 6, ht*wd});
      torch::Tensor E = torch::cat({Ei, Eij}, 0);

      SparseBlock S = schur_block(E, Q, w, ii_exp, jj_exp, kk_exp, t0, t1);
      dx = (A - S).solve(lm, ep);

      torch::Tensor ix = jj_exp - t0;
      torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

      EvT6x1_kernel<<<ix.size(0), THREADS>>>(
        E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

      dz = Q * (w - accum_cuda(dw, ii_exp, kx));

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);

      // update disparity maps
      disp_retr_kernel<<<kx.size(0), THREADS>>>(
        disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        kx.packed_accessor32<long,1,torch::RestrictPtrTraits>());
    }

  }

  return {dx, dz, dI};
}



torch::Tensor frame_distance_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj,
    const float beta,
    const int model_id)
{
  auto opts = poses.options();
  const int num = ii.size(0);

  torch::Tensor dist = torch::zeros({num}, opts);

  frame_distance_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    dist.packed_accessor32<float,1,torch::RestrictPtrTraits>(), beta, model_id);
  
  return dist;
}


std::vector<torch::Tensor> projmap_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor coords = torch::zeros({num, ht, wd, 3}, opts);
  torch::Tensor valid = torch::zeros({num, ht, wd, 1}, opts);

  projmap_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    valid.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return {coords, valid};
}


torch::Tensor depth_filter_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ix,
    torch::Tensor thresh)
{
  const int num = ix.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor counter = torch::zeros({num, ht, wd}, disps.options());

  dim3 blocks(num, 6, NUM_BLOCKS(ht * wd));

  depth_filter_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    thresh.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    counter.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  return counter;
}


torch::Tensor iproj_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics)
{

  const int nm = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  auto opts = disps.options();
  torch::Tensor points = torch::zeros({nm, ht, wd, 3}, opts);

  dim3 blocks(nm, NUM_BLOCKS(ht * wd));

  iproj_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    points.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return points;

}
